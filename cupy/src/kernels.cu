#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "kernels.h"
#include "cu.h"

#define THREADS_PER_BLOCK 256 
#define NUM_STREAMS 10

__global__ void elem_mul_kernel(const float *in1, const float *in2, float *out, const size_t size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= size) { return; }
  out[i] = in1[i] * in2[i];
}

void elemwise_mul(const float *in1, const float *in2, float *out, const size_t size) {
  int num_blocks = size / THREADS_PER_BLOCK + (size % THREADS_PER_BLOCK > 0);
  elem_mul_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(in1, in2, out, size);
  hipDeviceSynchronize();
}
