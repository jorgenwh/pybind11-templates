#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>

#include "extension_functions.h"
#include "kernels.h"

void elementwise_multiply(long in_data1, long in_data2, long out_data, size_t size) {
  float *in1 = reinterpret_cast<float*>(in_data1);
  float *in2 = reinterpret_cast<float*>(in_data2);
  float *out = reinterpret_cast<float*>(out_data);

  elemwise_mul(in1, in2, out, size);
}
