#include "hip/hip_runtime.h"
#include "kernels.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cmath>

__global__ void increment_kernel(float *data, size_t size, float value) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= size) { return; }

  data[i] += value;
}

void increment_array(float *data, size_t size, float value) {
  int num_blocks = size / BLOCK_SIZE + (size % BLOCK_SIZE > 0);
  increment_kernel<<<num_blocks, BLOCK_SIZE>>>(data, size, value);
  hipDeviceSynchronize();
}

__global__ void sigmoid_kernel(float *data, size_t size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= size) { return; }

  data[i] = 1.0f / (1.0f + exp(-data[i]));
}

void sigmoid_array(float *data, size_t size) {
  int num_blocks = size / BLOCK_SIZE + (size % BLOCK_SIZE > 0);
  sigmoid_kernel<<<num_blocks, BLOCK_SIZE>>>(data, size);
  hipDeviceSynchronize();
}

__global__ void relu_kernel(float *data, size_t size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= size) { return; }

  data[i] = (data[i] > 0) ? data[i] : 0;
}

void relu_array(float *data, size_t size) {
  int num_blocks = size / BLOCK_SIZE + (size % BLOCK_SIZE > 0);
  relu_kernel<<<num_blocks, BLOCK_SIZE>>>(data, size);
  hipDeviceSynchronize();
}
