#include "hip/hip_runtime.h"
#include "foo_functions.h"
#include "kernels.h"

#include <cmath>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>

py::array_t<float> &cpu_increment(py::array_t<float> &array, float value) {
  size_t data_size = array.size();
  float *data_ptr = array.mutable_data();

  for (py::ssize_t i = 0; i < data_size; i++) {
    data_ptr[i] = data_ptr[i] + value;
  }

  return array;
}

py::array_t<float> &gpu_increment(py::array_t<float> &array, float value) {
  size_t data_size = array.size();
  float *data_ptr = array.mutable_data();

  float *d_data_ptr;
  hipMalloc(&d_data_ptr, sizeof(float)*data_size);
  hipMemcpy(d_data_ptr, data_ptr, sizeof(float)*data_size, hipMemcpyHostToDevice);

  increment_array(d_data_ptr, data_size, value);

  hipMemcpy(data_ptr, d_data_ptr, sizeof(float)*data_size, hipMemcpyDeviceToHost);
  hipFree(d_data_ptr);

  return array;
}

py::array_t<float> &cpu_sigmoid(py::array_t<float> &array) {
  size_t data_size = array.size();
  float *data_ptr = array.mutable_data();

  for (py::ssize_t i = 0; i < data_size; i++) {
    data_ptr[i] = 1.0f / (1.0f + exp(-data_ptr[i]));
  }

  return array;
}

py::array_t<float> &gpu_sigmoid(py::array_t<float> &array) {
  size_t data_size = array.size();
  float *data_ptr = array.mutable_data();

  float *d_data_ptr;
  hipMalloc(&d_data_ptr, sizeof(float)*data_size);
  hipMemcpy(d_data_ptr, data_ptr, sizeof(float)*data_size, hipMemcpyHostToDevice);

  sigmoid_array(d_data_ptr, data_size);

  hipMemcpy(data_ptr, d_data_ptr, sizeof(float)*data_size, hipMemcpyDeviceToHost);
  hipFree(d_data_ptr);

  return array;
}

py::array_t<float> &cpu_relu(py::array_t<float> &array) {
  size_t data_size = array.size();
  float *data_ptr = array.mutable_data();

  for (py::ssize_t i = 0; i < data_size; i++) {
    data_ptr[i] = (data_ptr[i] > 0) ? data_ptr[i] : 0;
  }

  return array;
}

py::array_t<float> &gpu_relu(py::array_t<float> &array) {
  size_t data_size = array.size();
  float *data_ptr = array.mutable_data();

  float *d_data_ptr;
  hipMalloc(&d_data_ptr, sizeof(float)*data_size);
  hipMemcpy(d_data_ptr, data_ptr, sizeof(float)*data_size, hipMemcpyHostToDevice);

  relu_array(d_data_ptr, data_size);

  hipMemcpy(data_ptr, d_data_ptr, sizeof(float)*data_size, hipMemcpyDeviceToHost);
  hipFree(d_data_ptr);

  return array;
}
