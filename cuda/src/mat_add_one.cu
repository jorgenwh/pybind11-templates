#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "mat_add_one.h"

__global__ void add_one_kernel(float *arr, int size) {
  int i = threadIdx.x;

  if (i < size) {
    arr[i] = arr[i] + 1;
  }
}

void mat_add_one(float *arr, int size) {
  float *d_arr;

  hipMalloc(&d_arr, sizeof(float)*size);
  hipMemcpy(d_arr, arr, sizeof(float)*size, hipMemcpyHostToDevice);

  <<<1, size>>>add_one_kernel(d_arr, size);
  hipDeviceSynchronize();

  hipMemcpy(arr, d_arr, sizeof(float)*size, hipMemcpyDeviceToHost);
  hipFree(d_arr);
}
