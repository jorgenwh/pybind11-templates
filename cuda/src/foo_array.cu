#include <iostream>
#include <cstring>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "foo_array.h"
#include "foo_functions.h"

FooArray::FooArray(int size) {
  size_m = size;
  on_device_m = false;

  h_data_m = new float[size_m];
  memset(h_data_m, 0, sizeof(float)*size_m);
}

FooArray::~FooArray() {
  if (on_device_m) {
    hipFree(d_data_m);
  }
  else {
    delete[] h_data_m;
  }
}

void FooArray::to_host() {
  if (!on_device_m) { return; }

  h_data_m = new float[size_m];
  hipMemcpy(h_data_m, d_data_m, sizeof(float)*size_m, hipMemcpyDeviceToHost);
  hipFree(d_data_m);

  on_device_m = false;
}

void FooArray::to_device() {
  if (on_device_m) { return; }

  hipMalloc(&d_data_m, sizeof(float)*size_m);
  hipMemcpy(d_data_m, h_data_m, sizeof(float)*size_m, hipMemcpyHostToDevice);
  delete[] h_data_m;

  on_device_m = true;
}

void FooArray::print() {
  std::cout << "FooArray([";
  for (int i = 0; i < size_m; i++) {
    std::cout << h_data_m[i];
    if (i < size_m-1) { std::cout << ", "; }
  }
  std::cout << "], " << (on_device_m ? "device" : "host") << ")" << std::endl;
}

void FooArray::increment() {
  if (!on_device_m) { return; }

  increment_array(d_data_m, size_m);
}
