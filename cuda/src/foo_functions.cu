#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "foo_functions.h"

void foo_message() {
  std::cout << "foo message" << std::endl;
}

__global__ void increment_kernel(float *arr, int size) {
  int i = threadIdx.x;
  arr[i] = arr[i] + 1;
}

void increment_array(float *arr, int size) {
  increment_kernel<<<1, size>>>(arr, size);
  hipDeviceSynchronize();
}
