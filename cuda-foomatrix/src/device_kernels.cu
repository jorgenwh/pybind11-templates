#include "hip/hip_runtime.h"
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "device_kernels.h"

#define BLOCK_SIZE 252

__global__ void add_int_kernel(const int *in, const int value, int *out, const int size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= size) { return; }
  out[i] = in[i] + value;
}

void device_add_int(const int *in, const int value, int *out, const int size) {
  int num_blocks = size / BLOCK_SIZE + (size % BLOCK_SIZE > 0);
  add_int_kernel<<<num_blocks, BLOCK_SIZE>>>(in, value, out, size);
  hipDeviceSynchronize();
}

__global__ void transpose_kernel(const int *in, int *out, const int in_rows, const int in_cols) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= in_rows*in_cols) { return; }

  int in_row = floor((float)i / in_cols);
  int in_col = i % in_cols;
  int out_row = in_col;
  int out_col = in_row;

  out[out_row*in_rows + out_col] = in[in_row*in_cols + in_col];
}

void device_transpose(const int *in, int *out, const int in_rows, const int in_cols) {
  int size = in_rows*in_cols;
  int num_blocks = size / BLOCK_SIZE + (size % BLOCK_SIZE > 0);
  transpose_kernel<<<num_blocks, BLOCK_SIZE>>>(in, out, in_rows, in_cols);
  hipDeviceSynchronize();
}
