#include <string>
#include <assert.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sstream>

#include "foo_matrix.h"
#include "host_kernels.h"
#include "device_kernels.h"

FooMatrix::FooMatrix(int rows, int cols) {
  data = new int[rows*cols];
  this->rows = rows;
  this->cols = cols;

  int count = 10;
  for (int r = 0; r < rows; r++) {
    for (int c = 0; c < cols; c++) {
      data[r*cols + c] = count;
      count++;
    }
  }

  on_device = false;
}

FooMatrix::~FooMatrix() {
  if (on_device) {
    hipFree(data);
  } else {
    delete[] data;
  }
}

std::string FooMatrix::to_string() const {
  std::stringstream ss;

  int *dptr;
  bool needs_free = false;
  if (on_device) {
    dptr = new int[rows*cols];
    hipMemcpy(dptr, data, rows*cols*sizeof(int), hipMemcpyDeviceToHost);
    needs_free = true;
  } else {
    dptr = data;
  }

  ss << "FooMatrix(";
  ss << "[";
  for (int r = 0; r < rows; r++) {
    if (r != 0) { ss << "           "; }
    ss << "[";
    for (int c = 0; c < cols; c++) {
      ss << dptr[r*cols + c];
      if (c < cols - 1) { ss << ", "; }
    }
    if (r == rows - 1) { ss << "]"; }
    ss << "]";
    if (r != rows - 1) { ss << "\n"; }
  }
  ss << ", device=" << (on_device ? "device" : "host") << ")";

  if (needs_free) {
    assert(on_device);
    delete[] dptr;
  }
  return ss.str();
}

void FooMatrix::to_host() {
  if (!on_device) { return; }

  int *h_ptr = new int[rows*cols];
  hipMemcpy(h_ptr, data, rows*cols*sizeof(int), hipMemcpyDeviceToHost);
  hipFree(data);
  data = h_ptr;

  on_device = false;
}

void FooMatrix::to_device() {
  if (on_device) { return; }

  int *cu_ptr;
  hipMalloc(&cu_ptr, rows*cols*sizeof(int));
  hipMemcpy(cu_ptr, data, rows*cols*sizeof(int), hipMemcpyHostToDevice);
  delete[] data;
  data = cu_ptr;

  on_device = true;
}

FooMatrix FooMatrix::operator+(const int value) const {
  FooMatrix ret;

  ret.rows = rows;
  ret.cols = cols;
  ret.on_device = on_device;

  if (on_device) {
    size_t size = rows*cols*sizeof(int);
    hipMalloc(&ret.data, size);
    device_add_int(data, value, ret.data, rows*cols);
  } else {
    ret.data = new int[rows*cols];
    host_add_int(data, value, ret.data, rows*cols);
  }

  return ret;
}

FooMatrix &FooMatrix::operator+=(const int value) {
  int size = rows*cols;
  for (int i = 0; i < size; i++) { 
    data[i] += value;
  }
  return *this;
}

FooMatrix FooMatrix::transpose() const {
  FooMatrix ret;

  ret.rows = cols;
  ret.cols = rows;
  ret.on_device = on_device;

  if (on_device) {
    size_t size = rows*cols*sizeof(int);
    hipMalloc(&ret.data, size);
    device_transpose(data, ret.data, rows, cols);
  } else {
    ret.data = new int[cols*rows];
    host_transpose(data, ret.data, rows, cols);
  }

  return ret;
}
